#include "hip/hip_runtime.h"
//#include <wb.h>
#include "/home/prof/wagner/ci853/labs/wb3.h" // use our lib instead (under construction)
//#include "/home/wagner/ci853/labs-achel/wb.h" // use our lib instead (under construction)

#include <string.h>

#define BLOCK_DIM 256

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = threadIdx.x+blockDim.x*blockIdx.x;
  if(i<len) out[i] = in1[i] + in2[i];
  //@done
}

__host__ void printResult (float *out, int len) {
  int i;
  printf("%d\n", len);
  for (i = 0; i < len; ++i) {
    printf("%.2f\n", out[i]);
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  args = wbArg_read(argc, argv);
  // show_args( args ); // debug

  wbTime_start(Generic, "Importing data and creating memory on host");

  hostInput1 =
      (float *)wbImport( wbArg_getInputFile(args, 0), &inputLength );
  hostInput2 =
      (float *)wbImport( wbArg_getInputFile(args, 1), &inputLength );
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  // compute the size of the vector in the memory
  int size = inputLength * sizeof(float);
  // malloc ((cast to void because the methods expected a pointer to void input parameter))
  hipError_t err = hipMalloc((void **) &deviceInput1, size);
  if (err  != hipSuccess)  {
    printf("%s in %s at line %d\n",hipGetErrorString(err), __FILE__,__LINE__);
    exit(EXIT_FAILURE);
  }
  err = hipMalloc((void **) &deviceInput2, size);
  if (err  != hipSuccess)  {
    printf("%s in %s at line %d\n",hipGetErrorString(err), __FILE__,__LINE__);
    exit(EXIT_FAILURE);
  }
  err = hipMalloc((void **) &deviceOutput, size);
  if (err  != hipSuccess)  {
    printf("%s in %s at line %d\n",hipGetErrorString(err), __FILE__,__LINE__);
    exit(EXIT_FAILURE);
  }
  //@done
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  // memcpy (destination, origin, size in memory, flag for orientation)
  hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
  // hipMemcpy(deviceOutput, hostOutput, size, hipMemcpyHostToDevice);
  // @done
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  int blockDim = BLOCK_DIM;
  int gridDim = (((inputLength-1)/blockDim)+1); // ceil(n/256.0);
  //@done
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<gridDim, blockDim>>> (deviceInput1, deviceInput2, deviceOutput, inputLength);
  //@done
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  // memcpy (destination, origin, size in memory, flag for orientation)
  // hipMemcpy(hostInput1, deviceInput1, size, hipMemcpyDeviceToHost);
  // hipMemcpy(hostInput2, deviceInput2, size, hipMemcpyDeviceToHost);
  hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
  // @done
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  //@done
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, inputLength);

  // printResult(hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
