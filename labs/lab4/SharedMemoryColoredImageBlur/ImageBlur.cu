#include "hip/hip_runtime.h"
//#include <wb.h>
#include "/home/prof/wagner/ci853/labs/wb4.h" // use our lib instead (under construction)


#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5
#define CHANNELS 3

#define GTX480   480
#define GTX680   680
#define GPU     GTX680
#if GPU == GTX480
  #define MP    15  // number of mutiprocessors (SMs) in GTX480
  #define GRID1 (MP*2) // GRID size
  #define NT1   768
#elif GPU == GTX680
  #define MP    8 // number of mutiprocessors (SMs) in GTX680
  #define GRID1 (MP*2)
  #define NT1    1024
#endif

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData;
  unsigned char *deviceInputImageData;
  unsigned char *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 1);

  inputImage = wbImport(inputImageFile);

  // The input image is in grayscale, so the number of channels
  // is 1
  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

  outputImage = wbImage_new(imageWidth, imageHeight, CHANNELS);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");
  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 DimGrid((imageWidth-1)/16 + 1, (imageHeight-1)/16+1, 1);
  dim3 DimBlock(16, 16, 1);
  imageBlur<<<DimGrid,DimBlock>>>(deviceInputImageData, deviceOutputImageData, imageWidth, imageHeight );

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);
  wbExport("blured.ppm", outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
