#include "hip/hip_runtime.h"
#include "wb4.h"
//#include "/home/prof/wagner/ci853/labs/wb4.h" // use our lib instead (under construction)

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5
#define CHANNELS 3

#define GTX480   480
#define GTX680   680
#define MYGPU     GTX680
#if MYGPU == GTX480
  #define MP    15  // number of mutiprocessors (SMs) in GTX480
  #define GRID1 (MP*2) // GRID size
  #define NT1   768
#elif MYGPU == GTX680
  #define MP    8 // number of mutiprocessors (SMs) in GTX680
  #define GRID1 (MP*2)
  #define NT1    1024
#endif


__global__ void rgb2uintKernelSHM (unsigned int* inputImage, unsigned int* outputImage, int imageHeight, int imageWidth) {
  __shared__ unsigned int sharedInputImage [NT1];
  unsigned char * ucharInputImage = ( unsigned char *) sharedInputImage;

  // unique thread index
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int indexAtImage;
  char r, g, b;

  // iterate over the tiles
  for (int i = 0; i < imageHeight*imageWidth/(float)NT1; ++i)
  {

    // a thread can be outside the image?
    indexAtImage = index+i*NT1;
    // (((inputLength-1)/blockDim)+1)
    if ( indexAtImage < (imageWidth*imageHeight*CHANNELS/4.0) + 1) {
      // load the chunk
      sharedInputImage[threadIdx.x] = inputImage[indexAtImage];
    }
    __syncthreads();

    if ( indexAtImage < imageWidth*imageHeight) {
     
      r = ucharInputImage[threadIdx.x*CHANNELS+0];
      g = ucharInputImage[threadIdx.x*CHANNELS+1];
      b = ucharInputImage[threadIdx.x*CHANNELS+2];

      // printf("[%d %d %d]\n", r, g, b);

      unsigned int v = ((unsigned int)r << 16) + ((unsigned int)g << 8) + (unsigned int)b;

      // printf("%u\n",v);

      outputImage[index+i*NT1] = v;
    }
    __syncthreads();
  }

}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile; // name of the file
  wbImage_t inputImage; // the image loaded 
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData; // the final image
  unsigned char *deviceInputImageData;
  unsigned char *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 1);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

  outputImage = wbImage_new(imageWidth, imageHeight, CHANNELS);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  /// TESTE
  hostInputImageData = (unsigned char*) malloc (sizeof(unsigned char) * 15);
  hostInputImageData[0] = 0;
  hostInputImageData[1] = 1;
  hostInputImageData[2] = 2; // 258
  hostInputImageData[3] = 3;
  hostInputImageData[4] = 4;
  hostInputImageData[5] = 5; // 197637
  hostInputImageData[6] = 6;
  hostInputImageData[7] = 7;
  hostInputImageData[8] = 8; // 395016
  hostInputImageData[9] = 9;
  hostInputImageData[10] = 10;
  hostInputImageData[11] = 11; // 592395
  hostInputImageData[12] = 0;
  hostInputImageData[13] = 1;
  hostInputImageData[14] = 2; // 258

  imageWidth  = 5;
  imageHeight = 1;
  /// TESTE
  
  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char) + 7);
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");
  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  
  unsigned int * deviceAuxInputImage = (unsigned int *) deviceInputImageData;
  unsigned int * deviceConvertedImage;
  unsigned int * hostConvertedImage;
  //vector = (float *)malloc( n * sizeof(float) ); 
  hostConvertedImage = (unsigned int *) malloc (imageWidth * imageHeight * CHANNELS * sizeof(unsigned int));

  hipMalloc((void **)&deviceConvertedImage,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned int));

  //(unsigned int* inputImage, unsigned int* outputImage, int imageHeight, int imageWidth) {
  rgb2uintKernelSHM<<<GRID1, NT1>>> (deviceAuxInputImage, deviceConvertedImage, imageHeight, imageWidth);

  hipMemcpy(hostConvertedImage, deviceConvertedImage,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned int),
             hipMemcpyDeviceToHost);

  for (int i = 0; i < imageWidth * imageHeight; ++i)
  {
    printf("%u\n", hostConvertedImage[i]);
  }

  // //@@ INSERT CODE HERE
  // dim3 DimGrid((imageWidth-1)/16 + 1, (imageHeight-1)/16+1, 1);
  // dim3 DimBlock(16, 16, 1);
  // imageBlur<<<DimGrid,DimBlock>>>(deviceInputImageData, deviceOutputImageData, imageWidth, imageHeight );

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);
  wbExport("blured.ppm", outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
