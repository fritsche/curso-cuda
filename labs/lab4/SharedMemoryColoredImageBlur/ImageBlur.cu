#include "hip/hip_runtime.h"
//#include <wb.h>
#include "/home/prof/wagner/ci853/labs/wb4.h" // use our lib instead (under construction)


#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5
#define CHANNELS 3

#define GTX480   480
#define GTX680   680
#define MYGPU     GTX680
#if MYGPU == GTX480
  #define MP    15  // number of mutiprocessors (SMs) in GTX480
  #define GRID1 (MP*2) // GRID size
  #define NT1   768
#elif MYGPU == GTX680
  #define MP    8 // number of mutiprocessors (SMs) in GTX680
  #define GRID1 (MP*2)
  #define NT1    1024
#endif


__global__ void rgb2uintKernelSHM (unsigned int* inputImage, unsigned int* outputImage, int imageHeight, int imageWidth) {
  __shared__ unsigned int sharedInputImage [NT1];
  unsigned char * ucharInputImage = ( unsigned char *) sharedInputImage;

  // the thread index
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  char r, g, b;

  // iterate over the tiles
  for (int i = 0; i < imageHeight*imageWidth/NT1; ++i)
  {
    // a thread can be outside the image?
    if (! index+i*NT1 > imageWidth*imageHeight) {
      // load the chunk
      sharedInputImage[index] = inputImage[index+i*NT1];
    }
    __syncthreads();


    r = ucharInputImage[index+0];
    g = ucharInputImage[index+1];
    b = ucharInputImage[index+2];
    unsigned int v = ((unsigned int)r << 16) + ((unsigned int)g << 8) + (unsigned int)b;

    outputImage[index+i*NT1] = v;

  }

}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile; // name of the file
  wbImage_t inputImage; // the image loaded 
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData; // the final image
  unsigned char *deviceInputImageData;
  unsigned char *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 1);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

  outputImage = wbImage_new(imageWidth, imageHeight, CHANNELS);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char) + 3);
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");
  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  
  unsigned int * deviceAuxInputImage = (unsigned int *) deviceInputImageData;
  unsigned int * deviceConvertedImage;
  unsigned int * hostConvertedImage;
  //vector = (float *)malloc( n * sizeof(float) ); 
  hostConvertedImage = (unsigned int *) malloc (imageWidth * imageHeight * CHANNELS * sizeof(unsigned int));

  hipMalloc((void **)&deviceConvertedImage,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned int));

  //(unsigned int* inputImage, unsigned int* outputImage, int imageHeight, int imageWidth) {
  rgb2uintKernelSHM<<<GRID1, NT1>>> (deviceAuxInputImage, deviceConvertedImage, imageHeight, imageWidth);

  hipMemcpy(hostConvertedImage, deviceConvertedImage,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char),
             hipMemcpyDeviceToHost);

  for (int i = 0; i < imageWidth * imageHeight * CHANNELS; ++i)
  {
    printf("%d\n", hostConvertedImage[i]);
  }

  // //@@ INSERT CODE HERE
  // dim3 DimGrid((imageWidth-1)/16 + 1, (imageHeight-1)/16+1, 1);
  // dim3 DimBlock(16, 16, 1);
  // imageBlur<<<DimGrid,DimBlock>>>(deviceInputImageData, deviceOutputImageData, imageWidth, imageHeight );

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * CHANNELS * sizeof(unsigned char),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);
  wbExport("blured.ppm", outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
