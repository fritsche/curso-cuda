#include "hip/hip_runtime.h"
//#include <wb.h>
#include "wb4.h" // use our lib instead (under construction)
//#include "/home/wagner/ci853/labs-achel/wb.h" // use our lib instead (under construction)


#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5
#define CHANNELS 3

#define GTX480 480
#define GTX680 680
#define GPU GTX680
#if GPU == GTX680
	#define MP 15
	#define GRID1 (MP*2)
 	#define NT1 768
#elif GPU == GTX680
	#define MP 8
	#define GRID1(MP*2)
  #define NT1 1024
#endif



 __global__ void rgb2uintKernelSHM(unsigned int * inputImage, unsigned int * outputImage,
   int height, int width){

    __shared__ unsigned int sm[NT1];
    __shared__ unsigned char *sm2;

    unsigned int i;
    int tx = threadIdx.x;
    int size = height * width;
    unsigned char r, g, b, r2;
    for(i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i+= gridDim.x * blockDim.x){
        /* code */

        sm[tx] = inputImage[i];
        
        __syncthreads();
        sm2 = (unsigned char *)sm;

        r = sm2[tx*CHANNELS+0];
        g = sm2[tx*CHANNELS+1];
        b = sm2[tx*CHANNELS+2];


        outputImage[i] = ((unsigned int)r << 16) + ((unsigned int)g << 8) + (unsigned int) b;
        __syncthreads();
      }
    }

 __global__ void uint2rgbKernelSHM(unsigned int * inputImage, unsigned int * outputImage,
   int height, int width){

    // __shared__ unsigned int sm[NT1];
    // __shared__ unsigned char *sm2;
  unsigned char *sm2;

    unsigned int i;
    int tx = threadIdx.x;
    int size = height * width;
    unsigned char r, g, b, r2;
    unsigned char * outputImageChar = (unsigned char *) outputImage;

    for(i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i+= gridDim.x * blockDim.x){
        /* code */

        // sm[tx] = inputImage[i];
        
        //__syncthreads();
        
        sm2 = (unsigned char *) inputImage;

        r = sm2[tx*4+0];
        g = sm2[tx*4+1];
        b = sm2[tx*4+2];

        outputImageChar[i+0] = b;
        outputImageChar[i+1] = g;
        outputImageChar[i+2] = r;

        printf("alexandre: [%d %d %d]\n", outputImageChar[i+0], outputImageChar[i+1], outputImageChar[i+2] );

        //__syncthreads();
      }
    }




int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  //blur
  //unsigned char *hostInputImageData;
  //unsigned char *hostOutputImageData;
  //unsigned char *deviceInputImageData;
  //unsigned char *deviceOutputImageData;
  // rgb to urgb
  unsigned int *deviceUrgbInput;
  unsigned char *hostUrgbInput;
  unsigned int *deviceUrgbOutput;
  unsigned int *hostUrgbOutput;
  //TESTE
  unsigned char *hostInputImageData;
  unsigned int * hostConvertedImage;

  args = wbArg_read(argc, argv); /* parse the input arguments */
  inputImageFile = wbArg_getInputFile(args, 1);
  inputImage = wbImport(inputImageFile);
  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

  outputImage = wbImage_new(imageWidth, imageHeight, CHANNELS);

  hostUrgbInput  = wbImage_getData(inputImage);
  hostUrgbOutput = (unsigned int *) malloc(imageWidth * imageHeight * sizeof(unsigned int));
  //TESTE
  hostInputImageData = (unsigned char*) malloc (sizeof(unsigned char) * 15);
  hostInputImageData[0] = 0;
  hostInputImageData[1] = 1;
  hostInputImageData[2] = 2;
  hostInputImageData[3] = 3;
  hostInputImageData[4] = 4;
  hostInputImageData[5] = 5;
  hostInputImageData[6] = 6;
  hostInputImageData[7] = 7;
  hostInputImageData[8] = 8;
  hostInputImageData[9] = 9;
  hostInputImageData[10] = 10;
  hostInputImageData[11] = 11;
  hostInputImageData[12] = 0;
  hostInputImageData[13] = 1;
  hostInputImageData[14] = 2;

  imageWidth  = 5;
  imageHeight = 1;
  //TEste
  //allocating memory first kernel
  hipMalloc((void **)&deviceUrgbInput,
             imageWidth * imageHeight * sizeof(unsigned int));
  hipMalloc((void **)&deviceUrgbOutput,
             imageWidth * imageHeight * sizeof(unsigned int));
  //copying data to GPU
  hipMemcpy(deviceUrgbInput, hostInputImageData,
             imageWidth * imageHeight * sizeof(unsigned int),
             hipMemcpyHostToDevice);

  //lauching kernell
  hostConvertedImage = (unsigned int *) malloc (imageWidth * imageHeight * sizeof(unsigned char));
  // for (int i = 0; i < imageWidth * imageHeight * CHANNELS; ++i)
  // {
  //    printf("%u \n", hostConvertedImage[i]);
  //  }

  rgb2uintKernelSHM<<<GRID1,NT1>>>(deviceUrgbInput, deviceUrgbOutput, imageHeight, imageWidth);
  //copy data from gpu
  hipMemcpy(hostConvertedImage, deviceUrgbOutput,
   imageWidth * imageHeight * sizeof(unsigned int),hipMemcpyDeviceToHost);

   printf("chegou aqui\n");
   for (int i = 0; i < imageWidth * imageHeight; ++i)
   {
      printf("%u\n", hostConvertedImage[i]);
    }


  uint2rgbKernelSHM<<<GRID1,NT1>>>(deviceUrgbOutput, deviceUrgbInput, imageHeight, imageWidth);


  unsigned char * saida;

  hipMemcpy(saida, deviceUrgbInput,
    imageWidth * imageHeight * CHANNELS * sizeof(unsigned char), hipMemcpyDeviceToHost);

   printf("chegou aqui\n");
   for (int i = 0; i < imageWidth * imageHeight * CHANNELS; ++i)
   {
      printf("saida[%d]=%u\n", i, saida[i]);
    }

  //wbExport("blurred.ppm", hostUrgbOutput);

/*  wbSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);*/

  return 0;
}
