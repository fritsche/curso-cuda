#include "hip/hip_runtime.h"
//#include <wb.h>
#include "/home/prof/wagner/ci853/labs/wb3.h" // use our lib instead (under construction)
//#include "/home/wagner/ci853/labs-achel/wb.h" // use our lib instead (under construction)

#include <string.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ INSERT CODE HERE

# define CHANNELS 3

__global__ void colorConvert (unsigned char* grayImage, unsigned char* rgbImage, 
  int width, int height) {

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if ( x < width && y < height ) {
    // get 1D coordinate for the grayscale image
    int grayOffset = y*width + x;
    // one can think of the RGB image having
    // CHANNEL times columns than the gray scale image
    int rgbOffset = grayOffset *CHANNELS;
    unsigned char r =   rgbImage [rgbOffset]; 
    // red value for pixel 
    unsigned char g = rgbImage [rgbOffset + 1]; 
    // green value for pixel
    unsigned char b = rgbImage [rgbOffset + 2]; 
    // blue value for pixel

    grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
  }
}


// DONE

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData;
  unsigned char *deviceInputImageData;
  unsigned char *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */
  // show_args( args ); // debug

  inputImageFile = wbArg_getInputFile(args, 1);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  // For this lab the value is always 3
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(unsigned char));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 DimGrid((imageWidth-1)/16 + 1, (imageHeight-1)/16+1, 1);
  dim3 DimBlock(16, 16, 1);
  colorConvert<<<DimGrid,DimBlock>>>(deviceOutputImageData, deviceInputImageData, imageWidth, imageHeight );

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
