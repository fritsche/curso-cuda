#include "hip/hip_runtime.h"
//#include <wb.h>
#include "/home/prof/wagner/ci853/labs/wb3.h" // use our lib instead (under construction)
//#include "/home/wagner/ci853/labs-achel/wb.h" // use our lib instead (under construction)


#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define BLUR_SIZE 5

//@@ INSERT CODE HERE


__global__ void imageBlur (unsigned char* in, unsigned char* out, 
  int w, int h) {

  int Col = threadIdx.x + blockIdx.x * blockDim.x;
  int Row = threadIdx.y + blockIdx.y * blockDim.y;

  if ( Col < w && Row < h ) {
    
    int pixVal = 0;
    int pixels = 0;
    // Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
    for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
      for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {
        int curRow = Row + blurRow;
        int curCol = Col + blurCol;
        // Verify we have a valid image pixel
        if (curRow > -1 && curRow < h && curCol > -1 && curCol < w) { 
          pixVal += in[curRow * w + curCol];
          pixels++; 
          // Keep track of number of pixels in the accumulated total
        }
      }
    }
    // Write our new pixel value out
    out[Row * w + Col] = (unsigned char)(pixVal / pixels);
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  unsigned char *hostInputImageData;
  unsigned char *hostOutputImageData;
  unsigned char *deviceInputImageData;
  unsigned char *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 1);

  inputImage = wbImport(inputImageFile);

  // The input image is in grayscale, so the number of channels
  // is 1
  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);

  // Since the image is monochromatic, it only contains only one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * sizeof(unsigned char));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");
  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 DimGrid((imageWidth-1)/16 + 1, (imageHeight-1)/16+1, 1);
  dim3 DimBlock(16, 16, 1);
  imageBlur<<<DimGrid,DimBlock>>>(deviceInputImageData, deviceOutputImageData, imageWidth, imageHeight );

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(unsigned char),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
